// Frederich Stine EN.605.617
// Module 6 Assignment


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

/******************* CUDA Kernel Prototypes ********************/
__global__
void gpu_register_copy_arith (int32_t* blockOne, int32_t* resultBlock);
__global__
void gpu_register_arith (int32_t* blockOne, int32_t* resultBlock);
__global__
void gpu_global_arith (int32_t* blockOne, int32_t* resultBlock);

/******************* Core Function Prototypes ********************/
void run_all_arith (int op);

/******************* Global Variables ********************/
uint32_t threadCount = 0;
uint32_t blockSize = 0;
uint32_t numBlocks = 0;
uint32_t arrSizeBytes = 0;

/******************* Funtion definitions ********************/
// Main function
// This function takes in command line arguments and invokes the
// correct core functions
int main(int argc, char** argv) {
	// Prints out a help menu if not enough params are passed
	if (argc != 4) {
		printf("Call ./assignment {threadCount} {blockSize} {operation}\n");
		printf("Operations: \n");
		printf("    0: Copy to register\n");
		printf("    1: Register local variables\n");
		printf("    2: Global memory only\n");
		exit(0);
	}

	// Load the parameters from the command line
	threadCount = atoi(argv[1]);
	blockSize = atoi(argv[2]);
	numBlocks = (threadCount+(blockSize-1))/blockSize;
	arrSizeBytes = threadCount*sizeof(int32_t);
	int operation = atoi(argv[3]);

	// Run the kernel
	run_all_arith(operation);
}

void run_all_arith (int op) {
	int32_t *one, *result;
	int32_t *d_one, *d_result;

	// Allocated page locked  memory using standard c malloc function
	hipHostMalloc((void**)&one, arrSizeBytes, hipHostMallocDefault);
	hipHostMalloc((void**)&result, arrSizeBytes, hipHostMallocDefault);

	for(int i=0; i<1024; i++) {
		one[i] = i;
	}

	// Allocate memroy on the GPU for computation
	hipMalloc((void**)&d_one, arrSizeBytes);
	hipMalloc((void**)&d_result, arrSizeBytes);

	// Copy memory from host to GPU - pinned memory
	hipMemcpy(d_one, one, arrSizeBytes, hipMemcpyHostToDevice);

	// Run kernel
	switch (op) {
		case 0:
			gpu_register_copy_arith<<<numBlocks, blockSize>>>(d_one, d_result);
			break;
		case 1:
			gpu_register_arith<<<numBlocks, blockSize>>>(d_one, d_result);
			break;
		case 2:
			gpu_global_arith<<<numBlocks, blockSize>>>(d_one, d_result);
			break;
	}

	// Copy memory back from GPU to host
	hipMemcpy(result, d_result, arrSizeBytes, hipMemcpyDeviceToHost);
	
	// Free memory on GPU
	hipFree(d_one);
	hipFree(d_result);

	// Free pinned memory on host
	hipHostFree(one);
	hipHostFree(result);
}

__global__
void gpu_register_copy_arith (int32_t* blockOne, int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	int32_t localRegOne, localRegResult;
	localRegOne = blockOne[thread_idx];
	localRegResult = 0;

	for (int i=0; i<10000; i++) {
		localRegResult += localRegOne;
		localRegResult *= localRegOne;
		localRegResult -= localRegOne;
		localRegResult -= localRegOne;
	}

	// Copy back to global mem
	resultBlock[thread_idx] = localRegResult;
}


__global__
void gpu_register_arith (int32_t* blockOne, int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	int32_t localRegResult;
	localRegResult = 0;

	for (int i=0; i<10000; i++) {
		localRegResult += blockOne[thread_idx];
		localRegResult *= blockOne[thread_idx];
		localRegResult -= blockOne[thread_idx];
		localRegResult /= blockOne[thread_idx];
	}

	// Copy back to global mem
	resultBlock[thread_idx] = localRegResult;
}


__global__
void gpu_global_arith (int32_t* blockOne, int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	resultBlock[thread_idx] = 0;

	for (int i=0; i<10000; i++) {
		resultBlock[thread_idx] += blockOne[thread_idx];
		resultBlock[thread_idx] *= blockOne[thread_idx];
		resultBlock[thread_idx] -= blockOne[thread_idx];
		resultBlock[thread_idx] /= blockOne[thread_idx];
	}
}
