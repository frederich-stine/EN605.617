// Frederich Stine EN.605.617
// Module 8 Assignment Part 2


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <time.h>
#include <math.h>

#include <hipfft/hipfft.h>

/******************* Data Type Definitions ********************/
// Structure representing the header of a wav file
typedef struct {
	uint8_t ChunkID[4];
	uint32_t ChunkSize;
	uint8_t Format[4];
	uint8_t SubChunkID[4];
	uint32_t SubChunkSize;
	uint16_t AudioFormat;
	uint16_t NumChannels;
	uint32_t SampleRate;
	uint32_t ByteRate;
	uint16_t BlockAlign;
	uint16_t BitsPerSample;
	uint8_t SubChunk2ID[4];
	uint32_t SubChunk2Size;
} wavData;

/******************* CUDA Kernel Prototypes ********************/

/******************* Core Function Prototypes ********************/
// Function to run an FFT on a simple audio file
void runFFT (void);

/******************* Helper Function Prototypes ********************/

/******************* Global Variables ********************/
int numSamples, resultSize, resultSizeBytes;

/******************* Funtion definitions ********************/
int main (int argc, char** argv) {

	// Prints out a help menu if not enough params are passed
	if (argc != 2) {
		printf("Simple audio spectrum FFT example\n");
		printf("    Call ./assignment {FFT_SIZE} \n");
		exit(0);
	}

	// Set up data sizes
	numSamples = atoi(argv[1]);
	resultSize = (numSamples/2)+1;
	resultSizeBytes = resultSize*sizeof(hipfftComplex);

	// Run core function
	runFFT();
}

// Core function that reads from a wav file and runs a single 1d fft
void runFFT (void) {
	FILE* audioFh;
	wavData wavHeader;

	// Open WAV file for processing
	audioFh = fopen("440Hz_44100Hz_16bit_05sec.wav", "rb");
	fread(&wavHeader, 1, sizeof(wavData), audioFh);

	// Print out some data about the file
	printf("Sample rate: %d\n", wavHeader.SampleRate);
	printf("Bits per sample: %d\n", wavHeader.BitsPerSample);
	printf("Num channels: %d\n", wavHeader.NumChannels);

	// Prepare input data buffers
	hipfftReal* i_cu_buf;
	hipfftComplex* o_cu_buf;
	hipHostMalloc((void**)&i_cu_buf, \
			numSamples*sizeof(hipfftReal), hipHostMallocDefault);
	hipHostMalloc((void**)&o_cu_buf, \
			numSamples*sizeof(hipfftComplex), hipHostMallocDefault);

	// Read in samples from wav file
	int16_t input_buf[numSamples];
	fread(input_buf, numSamples*2, 1, audioFh);

	// Convert wav file to float values
	for(int i=0; i<numSamples; i++) {
		i_cu_buf[i] = (hipfftReal)input_buf[i];
		//printf("Input value: %f\n", i_cu_buf[i]);
	}

	// Allocate device data
	hipfftReal* d_i_cu_buf;
	hipfftComplex* d_o_cu_buf;

	hipMalloc((void **)&d_i_cu_buf, \
			numSamples*sizeof(hipfftReal));
	hipMalloc((void **)&d_o_cu_buf, resultSizeBytes);

	// Copy input to device
	hipMemcpy(d_i_cu_buf, i_cu_buf, \
			numSamples*sizeof(hipfftReal),\
			hipMemcpyHostToDevice);

	// Create cufft 1d plan
	// R2C conversion with 1 batch size
	hipfftHandle plan;
	hipfftPlan1d(&plan, numSamples, HIPFFT_R2C, 1);

	// Execute the fft
	hipfftExecR2C(plan, d_i_cu_buf, d_o_cu_buf);

	// Copy result back from GPU
	hipMemcpy(o_cu_buf, d_o_cu_buf, \
			resultSizeBytes, hipMemcpyDeviceToHost);

	// Calculate magnitude
	float* d_o_magnitude = (float*) malloc (resultSize*sizeof(float));
	for (int i=0; i<resultSize; i++) {
		d_o_magnitude[i] = sqrt(pow(o_cu_buf[i].x, 2) + pow(o_cu_buf[i].y, 2));
	}

	// Calculate magnitude db
	float* d_o_magnitude_db = (float*) malloc (resultSize*sizeof(float));
	for (int i=0; i<resultSize; i++) {
		d_o_magnitude_db[i] = 20*log10(d_o_magnitude[i]);
	}

	// Print out information about the results
	for (int i=0; i<resultSize; i++) {
		float frequency = (float)i*(float)wavHeader.SampleRate/(float)numSamples;
		printf("FFT Result: Frequency: %f: C: %f: I: %f \n"\
				"    Magnitude: %f: Magnitude dB: %f\n", \
				frequency, o_cu_buf[i].x, o_cu_buf[i].y, \
				d_o_magnitude[i], d_o_magnitude_db[i]);
	}

	// Free all memory
	hipfftDestroy(plan);

	hipFree(d_i_cu_buf);
	hipFree(d_o_cu_buf);

	hipHostFree(i_cu_buf);
	hipHostFree(o_cu_buf);

	free(d_o_magnitude);
	free(d_o_magnitude_db);
}

