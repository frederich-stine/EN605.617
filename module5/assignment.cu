// Frederich Stine EN.605.617
// Module 5 Assignment


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

/******************* CUDA Kernel Prototypes ********************/
__global__
void gpu_all_arith_shared (int32_t* blockOne, int32_t* resultBlock);
__global__
void gpu_all_arith_shared_copy (int32_t* blockOne, int32_t* resultBlock);
__global__
void gpu_all_arith_const (int32_t* resultBlock);
__global__
void gpu_all_arith_only_const (int32_t* resultBlock);

/******************* Core Function Prototypes ********************/
void run_gpu_all_arith_shared (int op);
void run_gpu_arith_const_copy (void);
void run_gpu_arith_const_only (void);

/******************* Helper Function Prototypes ********************/
void print_blocks (int32_t* resultArr);

/******************* Global Variables ********************/

// Global variables used throughout file
uint32_t threadCount = 1024;
uint32_t blockSize = 0;
uint32_t numBlocks = 0;
uint32_t arrSizeBytes = 0;

__constant__ int32_t const_arr[1024];
__constant__ int32_t value1 = 0x01234567;
__constant__ int32_t value2 = 0x89ABCDEF;
__constant__ int32_t value3 = 0x02468ACE;
__constant__ int32_t value4 = 0x13579BDF;

/******************* Funtion definitions ********************/
// Main function
// This function takes in command line arguments and invokes the
// correct core functions
int main(int argc, char** argv) {
	// Prints out a help menu if not enough params are passed
	if (argc != 3) {
		printf("Call ./assignment {blockSize} {operation}\n");
		printf("Operations: \n");
		printf("    0: Copy to shared Memory\n");
		printf("    1: Shared memory for local\n");
		printf("    2: Copy to constant memory\n");
		printf("    3: Constant memory only\n");
		exit(0);
	}

	// Load the parameters from the command line
	blockSize = atoi(argv[1]);
	numBlocks = (threadCount+(blockSize-1))/blockSize;
	arrSizeBytes = threadCount*sizeof(int32_t);
	int operation = atoi(argv[2]);

	// Switch statement to call correct core function
	switch (operation) {
	case 0:
	case 1:
		run_gpu_all_arith_shared(operation);
		break;
	case 2:
		run_gpu_arith_const_copy();
		break;
	case 3:
		run_gpu_arith_const_only();
		break;
	default:
		printf("Incorrect operation specified: %d", operation);
		exit(0);
	}
}

void run_gpu_all_arith_shared (int op) {
	int32_t *one, *result;
	int32_t *d_one, *d_result;

	// Allocated page locked  memory using standard c malloc function
	hipHostMalloc((void**)&one, arrSizeBytes, hipHostMallocDefault);
	hipHostMalloc((void**)&result, arrSizeBytes, hipHostMallocDefault);

	// Initialize memory - general initialization
	for(int i=0; i<1024; i++) {
		one[i] = i;
	}
	
	// Allocate memroy on the GPU for computation
	hipMalloc((void**)&d_one, arrSizeBytes);
	hipMalloc((void**)&d_result, arrSizeBytes);

	// Copy memory from host to GPU - pinned memory
	hipMemcpy(d_one, one, arrSizeBytes, hipMemcpyHostToDevice);

	// Run kernel
	switch (op) {
		case 0:
			gpu_all_arith_shared<<<numBlocks, blockSize, blockSize*4>>>(d_one, d_result);
			break;
		case 1:
			gpu_all_arith_shared_copy<<<numBlocks, blockSize, blockSize*8>>>(d_one, d_result);
			break;
	}

	// Copy memory back from GPU to host
	hipMemcpy(result, d_result, arrSizeBytes, hipMemcpyDeviceToHost);
	
	print_blocks(result);

	// Free memory on GPU
	hipFree(d_one);
	hipFree(d_result);

	// Free pinned memory on host
	hipHostFree(one);
	hipHostFree(result);
}

void run_gpu_arith_const_copy () {
	int32_t *result, *one;
	int32_t *d_result;

	// Allocated page locked  memory using standard c malloc function
	one = (int32_t*)malloc(arrSizeBytes);
	hipHostMalloc((void**)&result, arrSizeBytes, hipHostMallocDefault);

	// Initialize memory - general initialization
	for(int i=0; i<1024; i++) {
		one[i] = i;
	}
	
	// Allocate memory on the GPU for computation
	hipMalloc((void**)&d_result, arrSizeBytes);

	// Copy memory from host to GPU - pinned memory
	hipMemcpyToSymbol(HIP_SYMBOL(const_arr), one, arrSizeBytes);

	// Run kernel
	gpu_all_arith_const<<<numBlocks, blockSize>>>(d_result);

	// Copy memory back from GPU to host
	hipMemcpy(result, d_result, arrSizeBytes, hipMemcpyDeviceToHost);

	print_blocks(result);

	// Free memory on GPU
	hipFree(d_result);

	// Free pinned memory on host
	free(one);
	hipHostFree(result);
}


void run_gpu_arith_const_only (void) {
	int32_t *result;
	int32_t *d_result;

	// Allocated page locked  memory using standard c malloc function
	hipHostMalloc((void**)&result, arrSizeBytes, hipHostMallocDefault);
	
	// Allocate memory on the GPU for computation
	hipMalloc((void**)&d_result, arrSizeBytes);

	// Run kernel
	gpu_all_arith_only_const<<<numBlocks, blockSize>>>(d_result);

	// Copy memory back from GPU to host
	hipMemcpy(result, d_result, arrSizeBytes, hipMemcpyDeviceToHost);

	// Print out result
	print_blocks(result);

	// Free memory on GPU
	hipFree(d_result);

	// Free pinned memory on host
	hipHostFree(result);
}

__global__
void gpu_all_arith_shared (int32_t* blockOne, int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int32_t s[];

	s[threadIdx.x] = 0;
	s[threadIdx.x] += blockOne[thread_idx];
	s[threadIdx.x] *= blockOne[thread_idx];
	s[threadIdx.x] -= blockOne[thread_idx];
	s[threadIdx.x] /= blockOne[thread_idx];
	
	resultBlock[thread_idx] = s[threadIdx.x];
}

__global__
void gpu_all_arith_shared_copy (int32_t* blockOne, int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	extern __shared__ int32_t s[];

	// Copy to shared memory
	s[threadIdx.x] = blockOne[thread_idx];

	__syncthreads();

	// Execute
	s[threadIdx.x+blockDim.x] = 0;
	s[threadIdx.x+blockDim.x] += s[threadIdx.x];
	s[threadIdx.x+blockDim.x] *= s[threadIdx.x];
	s[threadIdx.x+blockDim.x] -= s[threadIdx.x];
	s[threadIdx.x+blockDim.x] /= s[threadIdx.x];
	
	__syncthreads();
	
	// Copy back to global
	resultBlock[thread_idx] = s[threadIdx.x+blockDim.x];
}

__global__
void gpu_all_arith_const (int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	resultBlock[thread_idx] = 0;
	resultBlock[thread_idx] += const_arr[thread_idx];
	resultBlock[thread_idx] *= const_arr[thread_idx];
	resultBlock[thread_idx] -= const_arr[thread_idx];
	resultBlock[thread_idx] /= const_arr[thread_idx];
}

__global__
void gpu_all_arith_only_const (int32_t* resultBlock) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	resultBlock[thread_idx] = 0;
	resultBlock[thread_idx] += value1;
	resultBlock[thread_idx] *= value2;
	resultBlock[thread_idx] -= value3;
	resultBlock[thread_idx] /= value4;
}

// Print helper function
// Prints all of the data in the array ordered in blocks
void print_blocks (int32_t* resultArr) {
	for (int i=0; i<numBlocks; i++) {
		printf("B%-2d ", i);
	}
	printf("\n");

	for (int i=0; i<blockSize; i++) {
		for (int x=0; x<numBlocks; x++) {
			printf("%-3d ", resultArr[i + (x*blockSize)]);
		}
		printf("\n");
	}
}
